#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <thrust/gather.h>
#include <cmath>
#include <glm/glm.hpp>
#include "utilityCore.hpp"
#include "kernel.h"

// LOOK-2.1 potentially useful for doing grid-based neighbor search
#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

/**
* Check for CUDA errors; print and exit if there was a problem.
*/
void checkCUDAError(const char *msg, int line = -1) {
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    if (line >= 0) {
      fprintf(stderr, "Line %d: ", line);
    }
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}


/*****************
* Configuration *
*****************/

/*! Block size used for CUDA kernel launch. */
#define blockSize 128

// LOOK-1.2 Parameters for the boids algorithm.
// These worked well in our reference implementation.
#define rule1Distance 5.0f
#define rule2Distance 3.0f
#define rule3Distance 5.0f

#define rule1Scale 0.01f
#define rule2Scale 0.1f
#define rule3Scale 0.1f

#define maxSpeed 1.0f

/*! Size of the starting area in simulation space. */
#define scene_scale 100.0f

/***********************************************
* Kernel state (pointers are device pointers) *
***********************************************/

int numObjects;
dim3 threadsPerBlock(blockSize);

// LOOK-1.2 - These buffers are here to hold all your boid information.
// These get allocated for you in Boids::initSimulation.
// Consider why you would need two velocity buffers in a simulation where each
// boid cares about its neighbors' velocities.
// These are called ping-pong buffers.
glm::vec3 *dev_pos;
glm::vec3 *dev_vel1;
glm::vec3 *dev_vel2;

// LOOK-2.1 - these are NOT allocated for you. You'll have to set up the thrust
// pointers on your own too.

// For efficient sorting and the uniform grid. These should always be parallel.
int *dev_particleArrayIndices; // What index in dev_pos and dev_velX represents this particle?
int *dev_particleGridIndices; // What grid cell is this particle in?
// needed for use with thrust
thrust::device_ptr<int> dev_thrust_particleArrayIndices;
thrust::device_ptr<int> dev_thrust_particleGridIndices;

int *dev_gridCellStartIndices; // What part of dev_particleArrayIndices belongs
int *dev_gridCellEndIndices;   // to this cell?

// TODO-2.3 - consider what additional buffers you might need to reshuffle
// the position and velocity data to be coherent within cells.
// thrust::device_ptr<glm::vec3> dev_thrust_pos;
// thrust::device_ptr<glm::vec3> dev_thrust_vel;
glm::vec3 *dev_pos_gather;
glm::vec3 *dev_vel_gather;


// LOOK-2.1 - Grid parameters based on simulation parameters.
// These are automatically computed for you in Boids::initSimulation
int gridCellCount;
int gridSideCount;
float gridCellWidth;
float gridInverseCellWidth; // inverse of gridCellWidth
glm::vec3 gridMinimum;

/******************
* initSimulation *
******************/

__host__ __device__ unsigned int hash(unsigned int a) {
  a = (a + 0x7ed55d16) + (a << 12);
  a = (a ^ 0xc761c23c) ^ (a >> 19);
  a = (a + 0x165667b1) + (a << 5);
  a = (a + 0xd3a2646c) ^ (a << 9);
  a = (a + 0xfd7046c5) + (a << 3);
  a = (a ^ 0xb55a4f09) ^ (a >> 16);
  return a;
}

/**
* LOOK-1.2 - this is a typical helper function for a CUDA kernel.
* Function for generating a random vec3.
*/
__host__ __device__ glm::vec3 generateRandomVec3(float time, int index) {
  thrust::default_random_engine rng(hash((int)(index * time)));
  thrust::uniform_real_distribution<float> unitDistrib(-1, 1);

  return glm::vec3((float)unitDistrib(rng), (float)unitDistrib(rng), (float)unitDistrib(rng));
}

/**
* LOOK-1.2 - This is a basic CUDA kernel.
* CUDA kernel for generating boids with a specified mass randomly around the star.
*/
__global__ void kernGenerateRandomPosArray(int time, int N, glm::vec3 * arr, float scale) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    glm::vec3 rand = generateRandomVec3(time, index);
    arr[index].x = scale * rand.x;
    arr[index].y = scale * rand.y;
    arr[index].z = scale * rand.z;
  }
}

/**
* Initialize memory, update some globals
*/
void Boids::initSimulation(int N) {
  numObjects = N;
  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  // LOOK-1.2 - This is basic CUDA memory management and error checking.
  // Don't forget to hipFree in  Boids::endSimulation.
  hipMalloc((void**)&dev_pos, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_pos failed!");

  hipMalloc((void**)&dev_vel1, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel1 failed!");

  hipMalloc((void**)&dev_vel2, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_vel2 failed!");

  // LOOK-1.2 - This is a typical CUDA kernel invocation.
  kernGenerateRandomPosArray<<<fullBlocksPerGrid, blockSize>>>(1, numObjects,
    dev_pos, scene_scale);
  checkCUDAErrorWithLine("kernGenerateRandomPosArray failed!");

  // LOOK-2.1 computing grid params
  gridCellWidth = 2.0f * std::max(std::max(rule1Distance, rule2Distance), rule3Distance); // double the neighbour distance
  int halfSideCount = (int)(scene_scale / gridCellWidth) + 1;
  gridSideCount = 2 * halfSideCount;

  gridCellCount = gridSideCount * gridSideCount * gridSideCount;
  gridInverseCellWidth = 1.0f / gridCellWidth;
  float halfGridWidth = gridCellWidth * halfSideCount;
  gridMinimum.x -= halfGridWidth;
  gridMinimum.y -= halfGridWidth;
  gridMinimum.z -= halfGridWidth;

  // TODO-2.1 TODO-2.3 - Allocate additional buffers here.
  hipMalloc((void**)&dev_particleArrayIndices, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_particleArrayIndices failed!");

  hipMalloc((void**)&dev_particleGridIndices, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_particleGridIndices failed!");

  hipMalloc((void**)&dev_gridCellStartIndices, gridCellCount * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellStartIndices failed!");

  hipMalloc((void**)&dev_gridCellEndIndices, gridCellCount * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_gridCellEndIndices failed!");

  hipMalloc((void**)&dev_pos_gather, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_thrust_pos_gather failed!");

  hipMalloc((void**)&dev_vel_gather, N * sizeof(glm::vec3));
  checkCUDAErrorWithLine("hipMalloc dev_thrust_vel_gather failed!");

  hipDeviceSynchronize();
}


/******************
* copyBoidsToVBO *
******************/

/**
* Copy the boid positions into the VBO so that they can be drawn by OpenGL.
*/
__global__ void kernCopyPositionsToVBO(int N, glm::vec3 *pos, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  float c_scale = -1.0f / s_scale;

  if (index < N) {
    vbo[4 * index + 0] = pos[index].x * c_scale;
    vbo[4 * index + 1] = pos[index].y * c_scale;
    vbo[4 * index + 2] = pos[index].z * c_scale;
    vbo[4 * index + 3] = 1.0f;
  }
}

__global__ void kernCopyVelocitiesToVBO(int N, glm::vec3 *vel, float *vbo, float s_scale) {
  int index = threadIdx.x + (blockIdx.x * blockDim.x);

  if (index < N) {
    vbo[4 * index + 0] = vel[index].x + 0.3f;
    vbo[4 * index + 1] = vel[index].y + 0.3f;
    vbo[4 * index + 2] = vel[index].z + 0.3f;
    vbo[4 * index + 3] = 1.0f;
  }
}

/**
* Wrapper for call to the kernCopyboidsToVBO CUDA kernel.
*/
void Boids::copyBoidsToVBO(float *vbodptr_positions, float *vbodptr_velocities) {
  dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);

  kernCopyPositionsToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_pos, vbodptr_positions, scene_scale);
  kernCopyVelocitiesToVBO << <fullBlocksPerGrid, blockSize >> >(numObjects, dev_vel1, vbodptr_velocities, scene_scale);

  checkCUDAErrorWithLine("copyBoidsToVBO failed!");

  hipDeviceSynchronize();
}


/******************
* stepSimulation *
******************/


/**
* LOOK-1.2 You can use this as a helper for kernUpdateVelocityBruteForce.
* __device__ code can be called from a __global__ context
* Compute the new velocity on the body with index `iSelf` due to the `N` boids
* in the `pos` and `vel` arrays.
*/
__device__ glm::vec3 computeVelocityChange(int N, int iSelf, const glm::vec3 *pos, const glm::vec3 *vel) {
    glm::vec3 perceived_center{}, seperate{}, perceived_v{}, v1{}, v2{}, v3{};
    int rule1_neighbors_count = 0, rule3_neighbors_count = 0;
    for (int i = 0; i < N; i++) {
        if (i == iSelf) {
            continue;
        }
        float distance = glm::distance(pos[i], pos[iSelf]);
        // Rule 1: boids fly towards their local perceived center of mass, which excludes themselves
        if (distance < rule1Distance) {
            perceived_center += pos[i];
            rule1_neighbors_count++;
        }
        // Rule 2: boids try to stay a distance d away from each other
        if (distance < rule2Distance) {
            seperate -= (pos[i] - pos[iSelf]);
        }
        // Rule 3: boids try to match the speed of surrounding boids
        if (distance < rule3Distance) {
            perceived_v += vel[i];
            rule3_neighbors_count++;
        }
    }
    // calculate velocity
    if (rule1_neighbors_count > 0) {
        perceived_center /= rule1_neighbors_count;
        v1 = (perceived_center - pos[iSelf]) * rule1Scale;
    }
    v2 = seperate * rule2Scale;
    if (rule3_neighbors_count > 0) {
        perceived_v /= rule3_neighbors_count;
        v3 = perceived_v * rule3Scale;
    }
    return vel[iSelf] + v1 + v2 + v3;
}

/**
* TODO-1.2 implement basic flocking
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdateVelocityBruteForce(int N, glm::vec3 *pos,
  glm::vec3 *vel1, glm::vec3 *vel2) {
  // Compute a new velocity based on pos and vel1
  // Clamp the speed
  // Record the new velocity into vel2. Question: why NOT vel1?
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= N) {
        return;
    }

    // clamp the new speed
    vel2[index] = computeVelocityChange(N, index, pos, vel1);
    if (glm::length(vel2[index]) > maxSpeed) {
        vel2[index] = glm::normalize(vel2[index]) * maxSpeed;
    }
}

/**
* LOOK-1.2 Since this is pretty trivial, we implemented it for you.
* For each of the `N` bodies, update its position based on its current velocity.
*/
__global__ void kernUpdatePos(int N, float dt, glm::vec3 *pos, glm::vec3 *vel) {
  // Update position by velocity
  int index = threadIdx.x + (blockIdx.x * blockDim.x);
  if (index >= N) {
    return;
  }
  glm::vec3 thisPos = pos[index];
  thisPos += vel[index] * dt;

  // Wrap the boids around so we don't lose them
  thisPos.x = thisPos.x < -scene_scale ? scene_scale : thisPos.x;
  thisPos.y = thisPos.y < -scene_scale ? scene_scale : thisPos.y;
  thisPos.z = thisPos.z < -scene_scale ? scene_scale : thisPos.z;

  thisPos.x = thisPos.x > scene_scale ? -scene_scale : thisPos.x;
  thisPos.y = thisPos.y > scene_scale ? -scene_scale : thisPos.y;
  thisPos.z = thisPos.z > scene_scale ? -scene_scale : thisPos.z;

  pos[index] = thisPos;
}

// LOOK-2.1 Consider this method of computing a 1D index from a 3D grid index.
// LOOK-2.3 Looking at this method, what would be the most memory efficient
//          order for iterating over neighboring grid cells?
//          for(x)
//            for(y)
//             for(z)? Or some other order?
__device__ int gridIndex3Dto1D(int x, int y, int z, int gridResolution) {
  return x + y * gridResolution + z * gridResolution * gridResolution;
}

__global__ void kernComputeIndices(int N, int gridResolution,
  glm::vec3 gridMin, float inverseCellWidth,
  glm::vec3 *pos, int *indices, int *gridIndices) {
    // TODO-2.1
    const auto index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) {
        return;
    }

    // - Label each boid with the index of its grid cell.
    glm::tvec3<unsigned int> grid_indices_3d = (pos[index] - gridMin) * inverseCellWidth;
    gridIndices[index] = gridIndex3Dto1D(grid_indices_3d.x, grid_indices_3d.y, grid_indices_3d.z, gridResolution);

    // - Set up a parallel array of integer indices as pointers to the actual
    //   boid data in pos and vel1/vel2
    indices[index] = index;
}

// LOOK-2.1 Consider how this could be useful for indicating that a cell
//          does not enclose any boids
__global__ void kernResetIntBuffer(int N, int *intBuffer, int value) {
  int index = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (index < N) {
    intBuffer[index] = value;
  }
}

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices,
  int *gridCellStartIndices, int *gridCellEndIndices) {
  // TODO-2.1
  // Identify the start point of each cell in the gridIndices array.
  // This is basically a parallel unrolling of a loop that goes
  // "this index doesn't match the one before it, must be a new cell!"
    const auto index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) {
        return;
    }
    const auto current_grid_index = particleGridIndices[index];
    if (index == 0) { // corner case
        gridCellStartIndices[current_grid_index] = 0;
        return;
    }
    const auto prev_grid_index = particleGridIndices[index - 1];
    if (prev_grid_index != current_grid_index) {
        gridCellEndIndices[prev_grid_index] = index - 1;
        gridCellStartIndices[current_grid_index] = index;
    }
    if (index == N - 1) { // corner case
        gridCellEndIndices[current_grid_index] = N - 1;
    }
}

__global__ void kernUpdateVelNeighborSearchScattered(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  int *particleArrayIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
    // TODO-2.1 - Update a boid's velocity using the uniform grid to reduce
    // the number of boids that need to be checked.
    const int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) {
        return;
    }

    glm::vec3 perceived_center{}, seperate{}, perceived_v{}, v1{}, v2{}, v3{};
    int rule1_neighbors_count = 0, rule3_neighbors_count = 0;

    // - Identify the grid cell that this particle is in
    const int x = std::round((pos[index].x - gridMin.x) * inverseCellWidth);
    const int y = std::round((pos[index].y - gridMin.y) * inverseCellWidth);
    const int z = std::round((pos[index].z - gridMin.z) * inverseCellWidth);
    const int x_start = imax(x - 1, 0);
    const int x_end = imin(x + 1, gridResolution - 1);
    const int y_start = imax(y - 1, 0);
    const int y_end = imin(y + 1, gridResolution - 1);
    const int z_start = imax(z - 1, 0);
    const int z_end = imin(z + 1, gridResolution - 1);
    for (int i = x_start; i < x_end; i++) {
        for (int j = y_start; j < y_end; j++) {
            for (int k = z_start; k < z_end; k++) {
                const auto grid = gridIndex3Dto1D(i, j, k, gridResolution);
                const int start = gridCellStartIndices[grid];
                const int end = gridCellEndIndices[grid];

                // - Identify which cells may contain neighbors. This isn't always 8.
                if (start == -1) { // no boids
                    continue;
                }

                // - For each cell, read the start/end indices in the boid pointer array.
                // - Access each boid in the cell and compute velocity change from
                //   the boids rules, if this boid is within the neighborhood distance.
                for (int p = start; p <= end; p++) {
                    const int b = particleArrayIndices[p];
                    if (b == index) {
                        continue;
                    }
                    const float distance = glm::distance(pos[b], pos[index]);
                    // rule 1
                    if (distance < rule1Distance) {
                        perceived_center += pos[b];
                        rule1_neighbors_count++;
                    }
                    // rule 2
                    if (distance < rule2Distance) {
                        seperate -= (pos[b] - pos[index]);
                    }
                    // rule 3
                    if (distance < rule3Distance) {
                        perceived_v += vel1[b];
                        rule3_neighbors_count++;
                    }
                }
            }
        }
    }
    if (rule1_neighbors_count > 0) {
        perceived_center /= rule1_neighbors_count;
        v1 = (perceived_center - pos[index]) * rule1Scale;
    }
    v2 = seperate * rule2Scale;
    if (rule3_neighbors_count > 0) {
        perceived_v /= rule3_neighbors_count;
        v3 = perceived_v * rule3Scale;
    }
    vel2[index] = vel1[index] + v1 + v2 + v3;

    // - Clamp the speed change before putting the new speed in vel2
    if (glm::length(vel2[index]) > maxSpeed) {
        vel2[index] = glm::normalize(vel2[index]) * maxSpeed;
    }
}

__global__ void kernUpdateVelNeighborSearchCoherent(
  int N, int gridResolution, glm::vec3 gridMin,
  float inverseCellWidth, float cellWidth,
  int *gridCellStartIndices, int *gridCellEndIndices,
  glm::vec3 *pos, glm::vec3 *vel1, glm::vec3 *vel2) {
  // TODO-2.3 - This should be very similar to kernUpdateVelNeighborSearchScattered,
  // except with one less level of indirection.
    const int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index >= N) {
        return;
    }

    glm::vec3 perceived_center{}, seperate{}, perceived_v{}, v1{}, v2{}, v3{};
    int rule1_neighbors_count = 0, rule3_neighbors_count = 0;

  // This should expect gridCellStartIndices and gridCellEndIndices to refer
  // directly to pos and vel1.
  // - Identify the grid cell that this particle is in
    const int x = std::round((pos[index].x - gridMin.x) * inverseCellWidth);
    const int y = std::round((pos[index].y - gridMin.y) * inverseCellWidth);
    const int z = std::round((pos[index].z - gridMin.z) * inverseCellWidth);
    const int x_start = imax(x - 1, 0);
    const int x_end = imin(x + 1, gridResolution - 1);
    const int y_start = imax(y - 1, 0);
    const int y_end = imin(y + 1, gridResolution - 1);
    const int z_start = imax(z - 1, 0);
    const int z_end = imin(z + 1, gridResolution - 1);

    for (int k = z_start; k < z_end; k++) {
        for (int j = y_start; j < y_end; j++) {
            for (int i = x_start; i < x_end; i++) {
                const auto grid = gridIndex3Dto1D(i, j, k, gridResolution);
                const int start = gridCellStartIndices[grid];
                const int end = gridCellEndIndices[grid];

                // - Identify which cells may contain neighbors. This isn't always 8.
                if (start == -1) {
                    continue;
                }

                // - For each cell, read the start/end indices in the boid pointer array.
                //   DIFFERENCE: For best results, consider what order the cells should be
                //   checked in to maximize the memory benefits of reordering the boids data.
                for (int b = start; b <= end; b++) {
                    if (b == index) {
                        continue;
                    }
                    const float distance = glm::distance(pos[b], pos[index]);
                    // - Access each boid in the cell and compute velocity change from
                    //   the boids rules, if this boid is within the neighborhood distance.
                    // rule 1
                    if (distance < rule1Distance) {
                        perceived_center += pos[b];
                        rule1_neighbors_count++;
                    }
                    // rule 2
                    if (distance < rule2Distance) {
                        seperate -= (pos[b] - pos[index]);
                    }
                    // rule 3
                    if (distance < rule3Distance) {
                        perceived_v += vel1[b];
                        rule3_neighbors_count++;
                    }
                }
            }
        }
    }
    if (rule1_neighbors_count > 0) {
        perceived_center /= rule1_neighbors_count;
        v1 = (perceived_center - pos[index]) * rule1Scale;
    }
    v2 = seperate * rule2Scale;
    if (rule3_neighbors_count > 0) {
        perceived_v /= rule3_neighbors_count;
        v3 = perceived_v * rule3Scale;
    }
    vel2[index] = vel1[index] + v1 + v2 + v3;

    // - Clamp the speed change before putting the new speed in vel2
    if (glm::length(vel2[index]) > maxSpeed) {
        vel2[index] = glm::normalize(vel2[index]) * maxSpeed;
    }
}

/**
* Step the entire N-body simulation by `dt` seconds.
*/
void Boids::stepSimulationNaive(float dt) {
  // TODO-1.2 - use the kernels you wrote to step the simulation forward in time.
  // TODO-1.2 ping-pong the velocity buffers
    const dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    kernUpdateVelocityBruteForce<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_pos, dev_vel1, dev_vel2);
    std::swap(dev_vel1, dev_vel2);
    kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel1);
}

void Boids::stepSimulationScatteredGrid(float dt) {
    const dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    const dim3 grids_per_block((gridCellCount + blockSize - 1) / blockSize);
  // TODO-2.1
  // Uniform Grid Neighbor search using Thrust sort.
  // In Parallel:
  // - label each particle with its array index as well as its grid index.
  //   Use 2x width grids.
    kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
    dev_thrust_particleArrayIndices = thrust::device_pointer_cast(dev_particleArrayIndices);
    dev_thrust_particleGridIndices = thrust::device_pointer_cast(dev_particleGridIndices);
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
    kernResetIntBuffer << < grids_per_block, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1); // init all to -1
    kernResetIntBuffer << < grids_per_block, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);
    kernIdentifyCellStartEnd<<<fullBlocksPerGrid, blockSize>>>(numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

  // - Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchScattered<<<fullBlocksPerGrid, blockSize>>>(numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_particleArrayIndices, dev_pos, dev_vel1, dev_vel2);

  // - Update positions
    kernUpdatePos<<<fullBlocksPerGrid, blockSize>>>(numObjects, dt, dev_pos, dev_vel2);

  // - Ping-pong buffers as needed
    std::swap(dev_vel1, dev_vel2);
}

void Boids::stepSimulationCoherentGrid(float dt) {
    const dim3 fullBlocksPerGrid((numObjects + blockSize - 1) / blockSize);
    const dim3 grids_per_block((gridCellCount + blockSize - 1) / blockSize);

  // TODO-2.3 - start by copying Boids::stepSimulationNaiveGrid
  // Uniform Grid Neighbor search using Thrust sort on cell-coherent data.
  // In Parallel:
  // - Label each particle with its array index as well as its grid index.
  //   Use 2x width grids
    kernComputeIndices << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, dev_pos, dev_particleArrayIndices, dev_particleGridIndices);

  // - Unstable key sort using Thrust. A stable sort isn't necessary, but you
  //   are welcome to do a performance comparison.
    dev_thrust_particleArrayIndices = thrust::device_pointer_cast(dev_particleArrayIndices);
    dev_thrust_particleGridIndices = thrust::device_pointer_cast(dev_particleGridIndices);
    thrust::sort_by_key(dev_thrust_particleGridIndices, dev_thrust_particleGridIndices + numObjects, dev_thrust_particleArrayIndices);

  // - Naively unroll the loop for finding the start and end indices of each
  //   cell's data pointers in the array of boid indices
    kernResetIntBuffer << < grids_per_block, blockSize >> > (gridCellCount, dev_gridCellStartIndices, -1); // init all to -1
    kernResetIntBuffer << < grids_per_block, blockSize >> > (gridCellCount, dev_gridCellEndIndices, -1);
    kernIdentifyCellStartEnd << <fullBlocksPerGrid, blockSize >> > (numObjects, dev_particleGridIndices, dev_gridCellStartIndices, dev_gridCellEndIndices);

  // - BIG DIFFERENCE: use the rearranged array index buffer to reshuffle all
  //   the particle data in the simulation array.
  //   CONSIDER WHAT ADDITIONAL BUFFERS YOU NEED
    auto dev_thrust_pos = thrust::device_pointer_cast(dev_pos);
    auto dev_thrust_vel = thrust::device_pointer_cast(dev_vel1);
    auto dev_thrust_pos_gather = thrust::device_pointer_cast(dev_pos_gather);
    auto dev_thrust_vel_gather = thrust::device_pointer_cast(dev_vel_gather);
    thrust::gather(dev_thrust_particleArrayIndices, dev_thrust_particleArrayIndices + numObjects, dev_thrust_pos, dev_thrust_pos_gather);
    thrust::gather(dev_thrust_particleArrayIndices, dev_thrust_particleArrayIndices + numObjects, dev_thrust_vel, dev_thrust_vel_gather);

  // - Perform velocity updates using neighbor search
    kernUpdateVelNeighborSearchCoherent << <fullBlocksPerGrid, blockSize >> > (numObjects, gridSideCount, gridMinimum, gridInverseCellWidth, gridCellWidth, dev_gridCellStartIndices, dev_gridCellEndIndices, dev_pos_gather, dev_vel_gather, dev_vel2);

  // - Update positions
    kernUpdatePos << <fullBlocksPerGrid, blockSize >> > (numObjects, dt, dev_pos_gather, dev_vel2);

  // - Ping-pong buffers as needed. THIS MAY BE DIFFERENT FROM BEFORE.
    std::swap(dev_vel1, dev_vel2);
    std::swap(dev_pos, dev_pos_gather);
}

void Boids::endSimulation() {
  hipFree(dev_vel1);
  hipFree(dev_vel2);
  hipFree(dev_pos);

  // TODO-2.1 TODO-2.3 - Free any additional buffers here.
  hipFree(dev_particleArrayIndices);
  hipFree(dev_particleGridIndices);
  hipFree(dev_gridCellStartIndices);
  hipFree(dev_gridCellEndIndices);
  hipFree(dev_pos_gather);
  hipFree(dev_vel_gather);
}

void Boids::unitTest() {
  // LOOK-1.2 Feel free to write additional tests here.

  // test unstable sort
  int *dev_intKeys;
  int *dev_intValues;
  int N = 10;

  std::unique_ptr<int[]>intKeys{ new int[N] };
  std::unique_ptr<int[]>intValues{ new int[N] };

  intKeys[0] = 0; intValues[0] = 0;
  intKeys[1] = 1; intValues[1] = 1;
  intKeys[2] = 0; intValues[2] = 2;
  intKeys[3] = 3; intValues[3] = 3;
  intKeys[4] = 0; intValues[4] = 4;
  intKeys[5] = 2; intValues[5] = 5;
  intKeys[6] = 2; intValues[6] = 6;
  intKeys[7] = 0; intValues[7] = 7;
  intKeys[8] = 5; intValues[8] = 8;
  intKeys[9] = 6; intValues[9] = 9;

  hipMalloc((void**)&dev_intKeys, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intKeys failed!");

  hipMalloc((void**)&dev_intValues, N * sizeof(int));
  checkCUDAErrorWithLine("hipMalloc dev_intValues failed!");

  dim3 fullBlocksPerGrid((N + blockSize - 1) / blockSize);

  std::cout << "before unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // How to copy data to the GPU
  hipMemcpy(dev_intKeys, intKeys.get(), sizeof(int) * N, hipMemcpyHostToDevice);
  hipMemcpy(dev_intValues, intValues.get(), sizeof(int) * N, hipMemcpyHostToDevice);

  // Wrap device vectors in thrust iterators for use with thrust.
  thrust::device_ptr<int> dev_thrust_keys(dev_intKeys);
  thrust::device_ptr<int> dev_thrust_values(dev_intValues);
  // LOOK-2.1 Example for using thrust::sort_by_key
  thrust::sort_by_key(dev_thrust_keys, dev_thrust_keys + N, dev_thrust_values);

  // How to copy data back to the CPU side from the GPU
  hipMemcpy(intKeys.get(), dev_intKeys, sizeof(int) * N, hipMemcpyDeviceToHost);
  hipMemcpy(intValues.get(), dev_intValues, sizeof(int) * N, hipMemcpyDeviceToHost);
  checkCUDAErrorWithLine("memcpy back failed!");

  std::cout << "after unstable sort: " << std::endl;
  for (int i = 0; i < N; i++) {
    std::cout << "  key: " << intKeys[i];
    std::cout << " value: " << intValues[i] << std::endl;
  }

  // cleanup
  hipFree(dev_intKeys);
  hipFree(dev_intValues);
  checkCUDAErrorWithLine("hipFree failed!");
  return;
}
